//general parts
#include <stdio.h>
#include <vector>
#include <memory>
#include <string.h>
#include <chrono>
#include <thread>
#include <iostream>

//CUDA parts
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define GROUP 1


void launch_benchmark_cuFFT_double(bool file_output, FILE* output)
{

	const int num_runs = 3;
	if (file_output)
		fprintf(output, "1 - cuFFT FFT + iFFT C2C benchmark 1D batched in double precision\n");
	printf("1 - cuFFT FFT + iFFT C2C benchmark 1D batched in double precision\n");
	double benchmark_result[2] = { 0,0 };//averaged result = sum(system_size/iteration_time)/num_benchmark_samples
	hipfftDoubleComplex* inputC = (hipfftDoubleComplex*)malloc((uint64_t)sizeof(hipfftDoubleComplex) *pow(2, 27));
	for (uint64_t i = 0; i <pow(2, 27); i++) {
		inputC[i].x = 2 * ((double)rand()) / RAND_MAX - 1.0;
		inputC[i].y = 2 * ((double)rand()) / RAND_MAX - 1.0;
	}
	for (int n = 0; n < 21; n++) {
		double run_time[num_runs][2];
		for (int r = 0; r < num_runs; r++) {
			hipfftHandle planZ2Z;
			hipfftDoubleComplex* dataC;

			uint32_t dims[3];
			dims[0] = 32 * pow(2, n); //Multidimensional FFT dimensions sizes (default 1). For best performance (and stability), order dimensions in descendant size order as: x>y>z.   
			if (n == 0) dims[0] = 2048;
			dims[1] = 64 * 32 * pow(2, 15) / dims[0];
			//dims[1] = (dims[1] > 32768) ? 32768 : dims[1];
			if (dims[1] == 0) dims[1] = 1;
			dims[2] = 1;
			hipMalloc((void**)&dataC, sizeof(hipfftDoubleComplex) * dims[0] * dims[1] * dims[2]);

			hipMemcpy(dataC, inputC, sizeof(hipfftDoubleComplex) * dims[0] * dims[1] * dims[2], hipMemcpyHostToDevice);
			if (hipGetLastError() != hipSuccess) {
				fprintf(stderr, "Cuda error: Failed to allocate\n");
				return;
			}
			uint64_t sizeCUDA;
			switch (1) {
			case 1:
				hipfftPlan1d(&planZ2Z, dims[0], HIPFFT_Z2Z, dims[1]);
				hipfftEstimate1d(dims[0], HIPFFT_Z2Z, 1, &sizeCUDA);
				break;
			case 2:
				hipfftPlan2d(&planZ2Z, dims[1], dims[0], HIPFFT_Z2Z);
				hipfftEstimate2d(dims[1], dims[0], HIPFFT_Z2Z, &sizeCUDA);
				break;
			case 3:
				hipfftPlan3d(&planZ2Z, dims[2], dims[1], dims[0], HIPFFT_Z2Z);
				hipfftEstimate3d(dims[2], dims[1], dims[0], HIPFFT_Z2Z, &sizeCUDA);
				break;
			}

			double totTime = 0;
			uint64_t cuBufferSize = sizeof(double) * 2 * dims[0] * dims[1] * dims[2];
			uint64_t batch = ((4096 * 1024.0 * 1024.0) / cuBufferSize > 1000) ? 1000 : (4096 * 1024.0 * 1024.0) / cuBufferSize ;
			if (batch == 0) batch = 1;
			auto timeSubmit = std::chrono::steady_clock::now();
			for (int i = 0; i < batch; i++) {

				hipfftExecZ2Z(planZ2Z, dataC, dataC, 1);
				hipfftExecZ2Z(planZ2Z, dataC, dataC, -1);
			}
			hipDeviceSynchronize();
			auto timeEnd = std::chrono::steady_clock::now();
			totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / batch;
			run_time[r][0] = totTime;
			if (n > 0) {
				if (r == num_runs - 1) {
					double std_error = 0;
					double avg_time = 0;
					for (uint32_t t = 0; t < num_runs; t++) {
						avg_time += run_time[t][0];
					}
					avg_time /= num_runs;
					for (uint32_t t = 0; t < num_runs; t++) {
						std_error += (run_time[t][0] - avg_time) * (run_time[t][0] - avg_time);
					}
					std_error = sqrt(std_error / num_runs);
					if (file_output)
						fprintf(output, "cuFFT System: %d %dx%d Buffer: %d MB avg_time_per_step: %0.3f ms std_error: %0.3f batch: %d benchmark: %d\n", (int)log2(dims[0]), dims[0], dims[1], cuBufferSize / 1024 / 1024, avg_time, std_error, batch, (int)(((double)cuBufferSize * sizeof(float) / sizeof(double) / 1024) / avg_time));

					printf("cuFFT System: %d %dx%d Buffer: %d MB avg_time_per_step: %0.3f ms std_error: %0.3f batch: %d benchmark: %d\n", (int)log2(dims[0]), dims[0], dims[1], cuBufferSize / 1024 / 1024, avg_time, std_error, batch, (int)(((double)cuBufferSize * sizeof(float) / sizeof(double) / 1024) / avg_time));
					benchmark_result[0] += ((double)cuBufferSize * sizeof(float)/sizeof(double)/ 1024) / avg_time;
				}

			}
			hipfftDestroy(planZ2Z);
			hipFree(dataC);
			hipDeviceSynchronize();
			//cufftDoubleComplex* output_cuFFT = (cufftDoubleComplex*)(malloc(sizeof(cufftDoubleComplex) * dims[0] * dims[1] * dims[2]));
			//cudaMemcpy(output_cuFFT, dataC, sizeof(cufftDoubleComplex) * dims[0] * dims[1] * dims[2], cudaMemcpyDeviceToHost);
			//cudaDeviceSynchronize();


		}
	}
	free(inputC);
	benchmark_result[0] /= (21 - 1);
	if (file_output)
		fprintf(output, "Benchmark score cuFFT: %d\n", (int)(benchmark_result[0]));
	printf("Benchmark score cuFFT: %d\n", (int)(benchmark_result[0]));

}

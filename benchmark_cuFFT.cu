#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hipfft/hipfft.h>

#define GROUP 1
#include <chrono>

int main()
{
    const int num_benchmark_samples_2D = 10;
    const int num_benchmark_samples_3D = 9;
    const int num_runs = 5;
    //cuFFT works best in when last dimension is the longest in R2C mode
    printf("First %d runs are a warmup\n", num_runs);
    int benchmark_dimensions_2D[num_benchmark_samples_2D][4] = { {1024, 1024, 1, 2},  {64, 64, 1, 2}, {256, 256, 1, 2}, {256, 1024, 1, 2}, {512, 512, 1, 2}, {1024, 1024, 1, 2}, {256, 4096, 1, 2}, {1024, 2048, 1, 2},{2048, 4096, 1, 2}, {4096, 4096, 1, 2} };
    int benchmark_dimensions_3D[num_benchmark_samples_3D][4] = { {32, 32, 32, 3}, {64, 64, 64, 3}, {32, 256, 256, 3}, {32, 256, 1024, 3}, {256, 256, 256, 3},  {8, 1024, 2048, 3},  {128, 512, 512, 3}, {256, 256, 2048, 3}, {8, 4096, 4096, 3}};
   
    //for 8k test
    /*const int num_benchmark_samples_2D = 6;
    const int num_benchmark_samples_3D = 3;
    const int num_runs = 5;
    int benchmark_dimensions_2D[num_benchmark_samples_2D][4] = { {1024, 1024, 1, 2}, {32, 8192, 1, 2}, {256, 8192, 1, 2}, {1024, 8192, 1, 2}, {4096, 8192, 1, 2}, {8192, 8192, 1, 2} };
    int benchmark_dimensions_3D[num_benchmark_samples_3D][4] = { {32, 32, 8192, 3}, {64, 256, 8192, 3}, {8, 1024, 8192, 3} };
    */
    //you can check this with arrays below
    //int benchmark_dimensions_2D[num_benchmark_samples_2D][4] = { {1024, 1024, 1, 2}, {32, 32, 1, 2}, {64, 64, 1, 2}, {256, 32, 1, 2}, {256, 256, 1, 2}, {1024, 256, 1, 2},{1024, 1024, 1, 2}, {4096, 256, 1, 2}, {4096, 2048, 1, 2}, {4096, 4096, 1, 2} };
    //int benchmark_dimensions_3D[num_benchmark_samples_3D][4] = { {32, 32, 32, 3}, {64, 64, 64, 3}, {256, 32, 32, 3}, {256, 256, 32, 3}, {256, 256, 256, 3}, {1024, 256, 32, 3}, {1024, 1024, 8, 3}, {2048, 1024, 8, 3}, {2048, 256, 256, 3}, {4096, 4096, 8, 3}, {4096, 4096, 32, 3} };
    double benchmark_result = 0;//averaged result = sum(system_size/iteration_time)/num_benchmark_samples

    for (int n = 0; n < num_benchmark_samples_2D; n++) {

        for (int r = 0; r < num_runs; r++) {
            hipfftHandle planR2C;
            hipfftHandle planC2R;
            hipfftComplex* dataC;
            hipfftReal* dataR;

            hipfftReal* inputReal;
            int dims[2] = { benchmark_dimensions_2D[n][0] , benchmark_dimensions_2D[n][1] };

            inputReal = (hipfftReal*)(malloc(sizeof(hipfftReal) * dims[0] * dims[1]));

            for (int j = 0; j < dims[1]; j++) {
                for (int i = 0; i < dims[0]; i++) {
                    inputReal[i + j * dims[0]] = i;
                }
            }
            hipMalloc((void**)&dataC, sizeof(hipfftComplex) * dims[0] * (dims[1] / 2 + 1));
            hipMalloc((void**)&dataR, sizeof(hipfftReal) * dims[0] * dims[1]);
            hipMemcpy(dataR, inputReal, sizeof(hipfftReal) * dims[0] * dims[1], hipMemcpyHostToDevice);
            if (hipGetLastError() != hipSuccess) {
                fprintf(stderr, "Cuda error: Failed to allocate\n");
                return;
            }

            if (hipfftPlanMany(&planC2R, 2, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_C2R, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create C2R plan\n");
                return;
            }
            if (hipfftPlanMany(&planR2C, 2, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_R2C, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create R2C plan\n");
                return;
            }

            double totTime = 0;
            int batch = ((512.0 * 1024.0 * 1024.0) / dims[0] / (dims[1] / 2 + 1) > 1000) ? 1000 : (512.0 * 1024.0 * 1024.0) / dims[0] / (dims[1] / 2 + 1);
            if (batch == 0) batch = 1;
            //batch *= 5;//makes result more smooth, takes longer time
            auto timeSubmit = std::chrono::steady_clock::now();
            hipDeviceSynchronize();
            for (int i = 0; i < batch; i++) {

                hipfftExecR2C(planR2C, dataR, dataC);
                hipfftExecC2R(planC2R, dataC, dataR);
            }
            hipDeviceSynchronize();
            auto timeEnd = std::chrono::steady_clock::now();
            totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / batch;

            printf("System: %dx%dx%d, run: %d, Buffer: %d MB, time per step: %0.3f ms, batch: %d\n", dims[1], dims[0], 1, r, (sizeof(hipfftReal) * dims[0] * dims[1] + sizeof(hipfftComplex) * dims[0] * (dims[1] / 2 + 1)) / 1024 / 1024, totTime, batch);
            int bufferSize = sizeof(float) * 2 * (dims[1] / 2 + 1) * dims[0];

            if (n > 0) benchmark_result += ((double)bufferSize / 1024) / totTime;

            /*cufftReal* output = (cufftReal*)(malloc(sizeof(cufftReal) * dims[0] * dims[1]));
            cudaMemcpy(output, dataR, sizeof(cufftReal) * dims[0] * dims[1], cudaMemcpyDeviceToHost);
            cudaDeviceSynchronize();
            for (int i = 0; i < 20; i++) {
                printf("%f %f\n", output[i] / (dims[0] * dims[1]), inputReal[i]);
            }*/
            hipfftDestroy(planR2C);
            hipFree(dataR);
            hipfftDestroy(planC2R);
            hipFree(dataC);
        }
    }
    for (int n = 0; n < num_benchmark_samples_3D; n++) {

        for (int r = 0; r < num_runs; r++) {
            hipfftHandle planR2C;
            hipfftHandle planC2R;
            hipfftComplex* dataC;
            hipfftReal* dataR;

            hipfftReal* inputReal;
            int dims[3] = { benchmark_dimensions_3D[n][0] , benchmark_dimensions_3D[n][1] , benchmark_dimensions_3D[n][2] };

            inputReal = (hipfftReal*)(malloc(sizeof(hipfftReal) * dims[0] * dims[1] * dims[2]));
            for (int k = 0; k < dims[2]; k++) {
                for (int j = 0; j < dims[1]; j++) {
                    for (int i = 0; i < dims[0]; i++) {
                        inputReal[i + j * dims[0] + k * dims[0] * dims[1]] = k;
                    }
                }
            }
            hipMalloc((void**)&dataC, sizeof(hipfftComplex) * dims[0] * dims[1] * (dims[2] / 2 + 1));
            hipMalloc((void**)&dataR, sizeof(hipfftReal) * dims[0] * dims[1] * dims[2]);
            hipMemcpy(dataR, inputReal, sizeof(hipfftReal) * dims[0] * dims[1] * dims[2], hipMemcpyHostToDevice);
            if (hipGetLastError() != hipSuccess) {
                fprintf(stderr, "Cuda error: Failed to allocate\n");
                return;
            }

            if (hipfftPlanMany(&planC2R, 3, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_C2R, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create C2R plan\n");
                return;
            }
            if (hipfftPlanMany(&planR2C, 3, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_R2C, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create R2C plan\n");
                return;
            }

            double totTime = 0;
            int batch = ((512.0 * 1024.0 * 1024.0) / dims[0] / dims[1] / (dims[2] / 2 + 1) > 1000) ? 1000 : (512.0 * 1024.0 * 1024.0) / dims[0] / dims[1] / (dims[2] / 2 + 1);
            if (batch == 0) batch = 1;
            //batch *= 5; //makes result more smooth, takes longer time
            auto timeSubmit = std::chrono::steady_clock::now();
            hipDeviceSynchronize();
            for (int i = 0; i < batch; i++) {

                hipfftExecR2C(planR2C, dataR, dataC);
                hipfftExecC2R(planC2R, dataC, dataR);

            }
            hipDeviceSynchronize();
            auto timeEnd = std::chrono::steady_clock::now();
            totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / batch;

            printf("System: %dx%dx%d, run: %d, Buffer: %d MB, time per step: %0.3f ms, batch: %d\n", dims[2], dims[1], dims[0], r, (sizeof(hipfftReal) * dims[0] * dims[1] * dims[2] + sizeof(hipfftComplex) * dims[0] * dims[1] * (dims[2] / 2 + 1)) / 1024 / 1024, totTime, batch);
            int bufferSize = sizeof(float) * 2 * (dims[2] / 2 + 1) * dims[1] * dims[0];
            benchmark_result += ((double)bufferSize / 1024) / totTime;

            hipfftDestroy(planR2C);
            hipFree(dataR);
            hipfftDestroy(planC2R);
            hipFree(dataC);
        }
    }
    benchmark_result /= ((num_benchmark_samples_3D + num_benchmark_samples_2D - 1) * num_runs);
    printf("Benchmark score: %d\n", (int)(benchmark_result));
}
